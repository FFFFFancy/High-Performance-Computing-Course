#include <iostream>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "hipblas.h"
using namespace std;

int main(int argc, char const *argv[])
{
    int M=atoi(argv[1]);
    int N=atoi(argv[2]);
    int K=atoi(argv[3]);

    // 分配 host 的内存空间
    double *host_A = (double*)malloc (N*M*sizeof(double));
    double *host_B = (double*)malloc (N*M*sizeof(double));
    double *host_C = (double*)malloc (M*M*sizeof(double));
    
    srand((unsigned)time(0));
    for (int i=0; i<N*M; i++) {
        host_A[i] = (double)rand() / (double)(RAND_MAX)*1e4;
        host_B[i] = (double)rand() / (double)(RAND_MAX)*1e4;
    }

    // 创建并初始化 CUBLAS 库对象
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double *device_A, *device_B, *device_C;
    // 分配 device 内存空间
    hipMalloc ((void**)&device_A, N * M * sizeof(double));
    hipMalloc ((void**)&device_B, N * M * sizeof(double));
    hipMalloc ((void**)&device_C, M * M * sizeof(double));

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // 将矩阵复制到显存中
    hipblasSetVector(N * M, sizeof(double), host_A, 1, device_A, 1);
    hipblasSetVector (N * M, sizeof(double), host_B, 1, device_B, 1);
    hipDeviceSynchronize();
    // 赋值alpha和beta，计算矩阵乘法
    double alpha=1;
    double beta=0;
    hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, M, N, &alpha, device_A, N, device_B, M, &beta, device_C, M);
    hipDeviceSynchronize();
    // 将结果复制回内存
    hipblasGetVector(M * M, sizeof(double), device_C, 1, host_C, 1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("matrixA: %dx%d  matrixB: %dx%d\n", M, N, N, K);
    printf("The time of CUBLAS: %f ms.\n", time);

    // 清理内存
    free (host_A);
    free (host_B);
    free (host_C);
    hipFree (device_A);
    hipFree (device_B);
    hipFree (device_C);
    // 释放 CUBLAS 库对象
    hipblasDestroy (handle);
    
    return 0;
}
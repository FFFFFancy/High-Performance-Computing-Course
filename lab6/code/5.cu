#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#define input_height 4096
#define input_width 4096
#define filter_size 3
#define stride 3
#define channels 3
#define padding 1

#define output_height (input_height + 2*padding - filter_size) / stride + 1
#define output_width (input_width + 2*padding - filter_size) / stride + 1

#define checkCUDNN(expression)                                     \
    {                                                              \
        hipdnnStatus_t status = (expression);                       \
        if (status != HIPDNN_STATUS_SUCCESS)                        \
        {                                                          \
            std::cerr << "Error on line " << __LINE__ << ": "      \
                      << hipdnnGetErrorString(status) << std::endl; \
            std::exit(EXIT_FAILURE);                               \
        }                                                          \
    }

int main(int argc, char const *argv[])
{
    hipdnnHandle_t cudnn;
	hipdnnCreate(&cudnn);
	
	float *image = (float *)malloc(input_height * input_width * channels * sizeof(float));
	// 初始化 input
	for (int i = 0; i < input_height * input_width * channels ; i++)
	{
		image[i] = (float)(rand() % 50)/100;
	}

    // input
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                          HIPDNN_TENSOR_NHWC,
                                          HIPDNN_DATA_FLOAT,
                                          1,3,
                                          input_height,
                                          input_width));

    // output
    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                          HIPDNN_TENSOR_NHWC,
                                          HIPDNN_DATA_FLOAT,
                                          1,1,
                                          output_height,
                                          output_width));

    // kernel
    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                          HIPDNN_DATA_FLOAT,
                                          HIPDNN_TENSOR_NCHW,
                                          1,3,3,3));

    // convolucion descriptor
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                               padding,padding,
                                               stride,stride,
                                               1,1,
                                               HIPDNN_CROSS_CORRELATION,
                                               HIPDNN_DATA_FLOAT));

    // algorithm
    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    checkCUDNN(
        hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                            input_descriptor,
                                            kernel_descriptor,
                                            convolution_descriptor,
                                            output_descriptor,
                                            HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                            0,
                                            &convolution_algorithm));

    // workspace size && allocate memory
    size_t workspace_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                       input_descriptor,
                                                       kernel_descriptor,
                                                       convolution_descriptor,
                                                       output_descriptor,
                                                       convolution_algorithm,
                                                       &workspace_bytes));

    
    void *d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);
    int image_bytes = 1 * 3 * input_height * input_width * sizeof(float);
    float *d_input{nullptr};
    hipMalloc(&d_input, image_bytes);
    hipMemcpy(d_input, image, image_bytes, hipMemcpyHostToDevice);
    float *d_output{nullptr};
    hipMalloc(&d_output, image_bytes);
    hipMemset(d_output, 0, image_bytes);

    // 初始化 kernel
    float h_kernel[3][1][3][3];
    for (int kernel = 0; kernel < 3; ++kernel)
    {
        for (int channel = 0; channel < 1; ++channel)
        {
            for (int row = 0; row < 3; ++row)
            {
                for (int column = 0; column < 3; ++column)
                {
                    h_kernel[kernel][channel][row][column] = (float)(rand() % 50)/100;
                }
            }
        }
    }

    float *d_kernel{nullptr};
    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

    // convolution
    const float alpha = 1, beta = 1;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       input_descriptor,
                                       d_input,
                                       kernel_descriptor,
                                       d_kernel,
                                       convolution_descriptor,
                                       convolution_algorithm,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       output_descriptor,
                                       d_output));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cuDNN CONVOLUTION TIME:     %fms\n", milliseconds / (3.0f * 1000.0f));
    
    float *h_output = (float *)malloc(image_bytes);
	hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);
    
    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
    free(image);
	free(h_output);
	free(d_workspace);
    
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);

    hipdnnDestroy(cudnn);
    
    return 0;
}